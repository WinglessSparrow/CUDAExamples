#include "DataParallel.cuh"

void DataParallel::executeCalculation(int *matrixA, int *matrixB, int *matrixC, int *oldBoard, int *newBoard, const size_t rows, const size_t cols)
{
   //allocation for game of life
   int *d_oldBoard;
   int *d_newBoard;

   size_t pitchOld;
   size_t pitchNew;

   //allocation for matrix multiplication
   int *d_matrixA;
   int *d_matrixB;
   int *d_matrixC;

   size_t pitchMA;
   size_t pitchMB;
   size_t pitchMC;

   //creating streams for memmory overlap
   hipStream_t stream1, stream2;

   hipStreamCreate(&stream1);
   hipStreamCreate(&stream2);

   //allocating all the necessary memory
   hipMallocPitch((void **)&d_oldBoard, (size_t *)&pitchOld, (size_t)cols * sizeof(int), (size_t)rows);
   hipMallocPitch((void **)&d_newBoard, (size_t *)&pitchNew, (size_t)cols * sizeof(int), (size_t)rows);
   hipMallocPitch((void **)&d_matrixA, (size_t *)&pitchMA, (size_t)cols * sizeof(int), (size_t)rows);
   hipMallocPitch((void **)&d_matrixB, (size_t *)&pitchMB, (size_t)cols * sizeof(int), (size_t)rows);
   hipMallocPitch((void **)&d_matrixC, (size_t *)&pitchMC, (size_t)cols * sizeof(int), (size_t)rows);

   //defining block and grid size
   dim3 grid(divideAndRound(rows, 16), divideAndRound(cols, 16));
   dim3 block(16, 16);

   //stream1
   hipMemcpy2DAsync(d_oldBoard, pitchOld, oldBoard, cols * sizeof(int), cols * sizeof(int), rows, hipMemcpyHostToDevice, stream1);

   numberAliveAround << <block, grid, 0, stream1 >> > (d_oldBoard, d_newBoard, cols, rows, pitchOld, pitchNew);
   determineNextState << <block, grid, 0, stream1 >> > (d_oldBoard, d_newBoard, cols, rows, pitchOld, pitchNew);

   //stream 2
   hipMemcpy2DAsync(d_matrixA, pitchMA, matrixA, cols * sizeof(int), cols * sizeof(int), rows, hipMemcpyHostToDevice, stream2);
   hipMemcpy2DAsync(d_matrixB, pitchMB, matrixB, cols * sizeof(int), cols * sizeof(int), rows, hipMemcpyHostToDevice, stream2);

   multiplyMatrix << <block, grid, 0, stream1 >> > (d_matrixA, d_matrixB, d_matrixC, cols, rows, pitchMA, pitchMB, pitchMC);

   hipMemcpy2DAsync(newBoard, cols * sizeof(int), d_newBoard, pitchNew, cols * sizeof(int), rows, hipMemcpyDeviceToHost, stream1);

   hipMemcpy2DAsync(matrixC, cols * sizeof(int), d_matrixC, pitchMC, cols * sizeof(int), rows, hipMemcpyDeviceToHost, stream2);

   hipDeviceSynchronize();

   //dealocating memory
   hipFree(d_oldBoard);
   hipFree(d_newBoard);
   hipFree(d_matrixA);
   hipFree(d_matrixB);
   hipFree(d_matrixC);

   hipStreamDestroy(stream1);
   hipStreamDestroy(stream2);
}

string DataParallel::getName()
{
   return string("Data parallel with overlap");
}

int DataParallel::divideAndRound(int numberElements, int blockSize)
{
   return ((numberElements % blockSize) != 0) ? (numberElements / blockSize + 1) : (numberElements / blockSize);
}