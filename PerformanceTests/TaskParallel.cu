#include "TaskParallel.cuh"

#include "DataParallel.cuh"

void TaskParallel::executeCalculation(int *matrixA, int *matrixB, int *matrixC, int *oldBoard, int *newBoard, const size_t rows, const size_t cols)
{

   const int CUDA_CORES = 1920;
   const int dataSize = rows * cols;
   const int numKernelCalls = (dataSize / (CUDA_CORES / 2) < 1) ? 1 : (dataSize / (CUDA_CORES / 2) + 1);

   // half of CUDA_COES per grid
   dim3 grid(2, 2);
   dim3 block(CUDA_CORES / 2 / 4 / 10, 10);

   //creating streams
   hipStream_t *streams = new hipStream_t[numKernelCalls * 2];
   for (int i = 0; i < numKernelCalls * 2; i++)
   {
      hipStreamCreate(&streams[i]);
   }

   //allocation for game of life
   int *d_oldBoard;
   int *d_newBoard;

   pitchesBoard boardPitches;

   //allocation for matrix multiplication
   int *d_matrixA;
   int *d_matrixB;
   int *d_matrixC;

   pitchesMatrix matrixPitches;

   //allocating memory
   hipMallocPitch((void **)&d_oldBoard, (size_t *)&boardPitches.pitchOld, (size_t)cols * sizeof(int), (size_t)rows);
   hipMallocPitch((void **)&d_newBoard, (size_t *)&boardPitches.pitchNew, (size_t)cols * sizeof(int), (size_t)rows);
   hipMallocPitch((void **)&d_matrixA, (size_t *)&matrixPitches.pitchMA, (size_t)cols * sizeof(int), (size_t)rows);
   hipMallocPitch((void **)&d_matrixB, (size_t *)&matrixPitches.pitchMB, (size_t)cols * sizeof(int), (size_t)rows);
   hipMallocPitch((void **)&d_matrixC, (size_t *)&matrixPitches.pitchMC, (size_t)cols * sizeof(int), (size_t)rows);

   hipMemcpy2D(d_oldBoard, boardPitches.pitchOld, oldBoard, cols * sizeof(int), cols * sizeof(int), rows, hipMemcpyHostToDevice);
   hipMemcpy2D(d_matrixA, matrixPitches.pitchMA, matrixA, cols * sizeof(int), cols * sizeof(int), rows, hipMemcpyHostToDevice);
   hipMemcpy2D(d_matrixB, matrixPitches.pitchMB, matrixB, cols * sizeof(int), cols * sizeof(int), rows, hipMemcpyHostToDevice);

   //starting all the streams
   int offset = 0;
   for (int i = 0; i < numKernelCalls; i += 2)
   {
      numberAliveAroundOffset << <block, grid, 0, streams[i] >> > (d_oldBoard, d_newBoard, cols, rows, boardPitches, offset * (CUDA_CORES / 2));
      determineNextStateOffset << <block, grid, 0, streams[i] >> > (d_oldBoard, d_newBoard, cols, rows, boardPitches, offset * (CUDA_CORES / 2));
      offset++;
      multiplyMatrixOffset << <block, grid, 0, streams[i + 1] >> > (d_matrixA, d_matrixB, d_matrixC, cols, rows, matrixPitches, offset * (CUDA_CORES / 2));
   }

   hipMemcpy2D(newBoard, cols * sizeof(int), d_newBoard, boardPitches.pitchNew, cols * sizeof(int), rows, hipMemcpyDeviceToHost);
   hipMemcpy2D(matrixC, cols * sizeof(int), d_matrixC, matrixPitches.pitchMC, cols * sizeof(int), rows, hipMemcpyDeviceToHost);


   hipFree(d_oldBoard);
   hipFree(d_newBoard);
   hipFree(d_matrixA);
   hipFree(d_matrixB);
   hipFree(d_matrixC);

   for (int i = 0; i < numKernelCalls * 2; i++)
   {
      hipStreamSynchronize(streams[i]);
      hipStreamDestroy(streams[i]);
   }

   hipDeviceSynchronize();

   delete[] streams;
}

string TaskParallel::getName()
{
   return string("Task Parallel");
}

int TaskParallel::divideAndRound(int numberElements, int blockSize)
{
   return ((numberElements % blockSize) != 0) ? (numberElements / blockSize + 1) : (numberElements / blockSize);
}
