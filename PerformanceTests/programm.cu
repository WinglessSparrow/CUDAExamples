#include "hip/hip_runtime.h"
﻿#include <stdio.h>
#include "Driver.h"

#include <vector>

using namespace std;


//tweking variables
#define NUM_ELEMENTS 4000
#define NUM_RUNS 300

void printTimers(vector<Timer> timers)
{
   for each (auto t in timers)
   {
      cout << "Test of " << t.getName() << " : Miliseconds: " << t.calcTimes().count() << "; Nanoseconds: " << t.calcTimesNano().count() << endl;
   }
}

__global__ void blankKernel()
{
}

int main()
{
   Driver driver;
   vector<Timer> timers;
   vector<TestBase *> tests;


   tests.push_back(new DataParallel);
   tests.push_back(new DataParallelNoOverlap());
   tests.push_back(new TaskParallel());

   //blank kernel, because first one always starts slower than the rest
   blankKernel << <1, 1 >> > ();
   hipDeviceSynchronize();

   cout << "Starting testing with " << NUM_ELEMENTS << " elements" << endl;

   for each (auto t in tests)
   {
      cout << "Test of: " << t->getName() << endl;
      timers.push_back(driver.runTest(NUM_ELEMENTS / 2, NUM_ELEMENTS / 2, NUM_RUNS, t));
   }


   printTimers(timers);

   return 0;
}

