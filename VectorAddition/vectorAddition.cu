#include "hip/hip_runtime.h"
﻿#include <hip/hip_runtime.h>
#include <>
#include <vector>
#include <iostream>
#include <chrono>


using std::vector;
using std::chrono::high_resolution_clock;
using std::chrono::duration_cast;
using std::chrono::duration;
using std::chrono::milliseconds;
using std::cout;
using std::endl;

//Kernel - the code that is beign run on the GPU
__global__ void vectorAdd1D(int *vectorA, int *vectorB, int *vectorOutput, int amountElements)
{
   //calculating the thread we are on
   int tid = (blockIdx.x * blockDim.x) + threadIdx.x;

   //making sure we are not out of bounds, because there might be more threads involved then the amount of elements
   if (tid < amountElements)
   {
      //calculating
      vectorOutput[tid] = vectorA[tid] + vectorB[tid];
   }
}

int main(void)
{
   constexpr int MAX_ELEMENTS = 1 << 4;
   constexpr int NUM_THREADS = 1 << 10;
   constexpr int NUM_BLOCKS = (MAX_ELEMENTS + NUM_THREADS - 1) / NUM_THREADS;

   int vectorSize = sizeof(int) * MAX_ELEMENTS;

   //size will be defined in the for loops
   vector<int> vectorA;
   vector<int> vectorB;
   //needs to be of defined size, because elemnts will be copied over pointers
   vector<int> vectorC(MAX_ELEMENTS);

   //generating random vectors
   for (int i = 0; i < MAX_ELEMENTS; i++) { vectorA.push_back(rand() % 100); }
   for (int i = 0; i < MAX_ELEMENTS; i++) { vectorB.push_back(rand() % 100); }

   for (int i = 0; i < MAX_ELEMENTS; i++)
   {
      cout << vectorA[i] << "+" << vectorB[i] << endl;
   }


   //pointers for GPU memory
   int *d_VectorA;
   int *d_VectorB;
   int *d_VectorC;

   auto timer1 = high_resolution_clock::now();
   //Allocating GPU memory
   hipMalloc(&d_VectorA, vectorSize);
   hipMalloc(&d_VectorB, vectorSize);
   hipMalloc(&d_VectorC, vectorSize);

   //Transfering data to from CPU to GPU
   hipMemcpy(d_VectorA, vectorA.data(), vectorSize, hipMemcpyHostToDevice);
   hipMemcpy(d_VectorB, vectorB.data(), vectorSize, hipMemcpyHostToDevice);

   //calling the function
   vectorAdd1D << <NUM_BLOCKS, NUM_THREADS >> > (d_VectorA, d_VectorB, d_VectorC, MAX_ELEMENTS);

   //retrieveing data from GPU
   hipMemcpy(vectorC.data(), d_VectorC, vectorSize, hipMemcpyDeviceToHost);

   auto timer2 = high_resolution_clock::now();

   auto ms_int = duration_cast<milliseconds>(timer2 - timer1);
   duration<double, std::milli> ms_double = timer2 - timer1;

   hipFree(d_VectorA);
   hipFree(d_VectorB);
   hipFree(d_VectorC);

   for (int i = 0; i < MAX_ELEMENTS; i++)
   {
      cout << vectorA[i] << "+" << vectorB[i] << "=" << vectorC[i] << endl;
   }

   std::cout << ms_int.count() << "ms\n";
   std::cout << ms_double.count() << "ms";

   std::cout << "end" << std::endl;

   return 1;
}