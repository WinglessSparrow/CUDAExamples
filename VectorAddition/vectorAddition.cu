#include "hip/hip_runtime.h"
﻿#include <hip/hip_runtime.h>
#include <>
#include <vector>
#include <iostream>
#include <chrono>


using std::vector;
using std::chrono::high_resolution_clock;
using std::chrono::duration_cast;
using std::chrono::duration;
using std::chrono::milliseconds;

//Kernel - the code that is beign run on the GPU
__global__ void vectorAdd1D(int *vectorA, int *vectorB, int *vectorOutput, int amountElements)
{
   //calculating the thread we are on
   int tid = (blockIdx.x * blockDim.x) + threadIdx.x;

   //making sure we are not out of bounds, because there might be more threads involved then the amount of elements
   if (tid < amountElements)
   {
      //calculating
      vectorOutput[tid] = vectorA[tid] + vectorB[tid];
   }
}

int main(void)
{
   constexpr int MAX_ELEMENTS = 1 << 16;
   constexpr int NUM_THREADS = 1 << 10;
   constexpr int NUM_BLOCKS = (MAX_ELEMENTS + NUM_THREADS - 1) / NUM_THREADS;

   int vectorSize = sizeof(int) * MAX_ELEMENTS;

   //size will be defined in the for loops
   vector<int> vectorA;
   vector<int> vectorB;
   //needs to be of defined size, because elemnts will be copied over pointers
   vector<int> vectorC(MAX_ELEMENTS);

   //generating random vectors
   for (int i = 0; i < MAX_ELEMENTS; i++) { vectorA.push_back(rand() % 100); }
   for (int i = 0; i < MAX_ELEMENTS; i++) { vectorB.push_back(rand() % 100); }

   //pointers for GPU memory
   int *gpuVectorA;
   int *gpuVectorB;
   int *gpuVectorC;

   auto timer1 = high_resolution_clock::now();
   //Allocating GPU memory
   hipMalloc(&gpuVectorA, vectorSize);
   hipMalloc(&gpuVectorB, vectorSize);
   hipMalloc(&gpuVectorC, vectorSize);

   //Transfering data to from CPU to GPU
   hipMemcpy(gpuVectorA, vectorA.data(), vectorSize, hipMemcpyHostToDevice);
   hipMemcpy(gpuVectorB, vectorB.data(), vectorSize, hipMemcpyHostToDevice);

   //calling the function
   vectorAdd1D << <NUM_BLOCKS, NUM_THREADS >> > (gpuVectorA, gpuVectorB, gpuVectorC, MAX_ELEMENTS);

   //retrieveing data from GPU
   hipMemcpy(vectorC.data(), gpuVectorC, vectorSize, hipMemcpyDeviceToHost);

   auto timer2 = high_resolution_clock::now();

   auto ms_int = duration_cast<milliseconds>(timer2 - timer1);
   duration<double, std::milli> ms_double = timer2 - timer1;

   hipFree(gpuVectorA);
   hipFree(gpuVectorB);
   hipFree(gpuVectorC);

   std::cout << ms_int.count() << "ms\n";
   std::cout << ms_double.count() << "ms";

   std::cout << "end" << std::endl;

   return 1;
}